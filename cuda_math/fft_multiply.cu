#include "hip/hip_runtime.h"
#include "fft_multiply.h"

#include "bigint.h"
#include "carry.h"
#include "memory.h"

#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include <hip/hip_complex.h>


#define FFT_BLOCK_SIZE (128)

__global__ void
split8(const unsigned int* __restrict__ in,
      hipDoubleComplex* __restrict__ out)
{
    int idx = (blockIdx.x*blockDim.x + threadIdx.x);
    
    out[idx*4].x = (double) (in[idx] & 0xff);
    out[idx*4].y = 0;
    
    out[idx*4+1].x = (double) ((in[idx] >> 8) & 0xff);
    out[idx*4+1].y = 0;
    
    out[idx*4+2].x = (double) ((in[idx] >> 16) & 0xff);
    out[idx*4+2].y = 0;
    
    out[idx*4+3].x = (double) ((in[idx] >> 24) & 0xff);
    out[idx*4+3].y = 0;
}

__global__ void
split4(const unsigned int* __restrict__ in,
      hipDoubleComplex* __restrict__ out)
{
    int idx = (blockIdx.x*blockDim.x + threadIdx.x);
    
    out[idx*8].x = (double) (in[idx] & 0xf);
    out[idx*8].y = 0;
    
    out[idx*8+1].x = (double) ((in[idx] >> 4) & 0xf);
    out[idx*8+1].y = 0;
    
    out[idx*8+2].x = (double) ((in[idx] >> 8) & 0xf);
    out[idx*8+2].y = 0;
    
    out[idx*8+3].x = (double) ((in[idx] >> 12) & 0xf);
    out[idx*8+3].y = 0;
    
    out[idx*8+4].x = (double) ((in[idx] >> 16) & 0xf);
    out[idx*8+4].y = 0;
    
    out[idx*8+5].x = (double) ((in[idx] >> 20) & 0xf);
    out[idx*8+5].y = 0;
    
    out[idx*8+6].x = (double) ((in[idx] >> 24) & 0xf);
    out[idx*8+6].y = 0;
    
    out[idx*8+7].x = (double) ((in[idx] >> 28) & 0xf);
    out[idx*8+7].y = 0;
}

__global__ void
split2(const unsigned int* __restrict__ in,
      hipDoubleComplex* __restrict__ out)
{
    int idx = (blockIdx.x*blockDim.x + threadIdx.x);
    
    out[idx*16].x = (double) (in[idx] & 0x3);
    out[idx*16].y = 0;
    
    out[idx*16+1].x = (double) ((in[idx] >> 2) & 0x3);
    out[idx*16+1].y = 0;
    
    out[idx*16+2].x = (double) ((in[idx] >> 4) & 0x3);
    out[idx*16+2].y = 0;
    
    out[idx*16+3].x = (double) ((in[idx] >> 6) & 0x3);
    out[idx*16+3].y = 0;
    
    out[idx*16+4].x = (double) ((in[idx] >> 8) & 0x3);
    out[idx*16+4].y = 0;
    
    out[idx*16+5].x = (double) ((in[idx] >> 10) & 0x3);
    out[idx*16+5].y = 0;
    
    out[idx*16+6].x = (double) ((in[idx] >> 12) & 0x3);
    out[idx*16+6].y = 0;
    
    out[idx*16+7].x = (double) ((in[idx] >> 14) & 0x3);
    out[idx*16+7].y = 0;
    
    out[idx*16+8].x = (double) ((in[idx] >> 16) & 0x3);
    out[idx*16+8].y = 0;
    
    out[idx*16+9].x = (double) ((in[idx] >> 18) & 0x3);
    out[idx*16+9].y = 0;
    
    out[idx*16+10].x = (double) ((in[idx] >> 20) & 0x3);
    out[idx*16+10].y = 0;
    
    out[idx*16+11].x = (double) ((in[idx] >> 22) & 0x3);
    out[idx*16+11].y = 0;
    
    out[idx*16+12].x = (double) ((in[idx] >> 24) & 0x3);
    out[idx*16+12].y = 0;
    
    out[idx*16+13].x = (double) ((in[idx] >> 26) & 0x3);
    out[idx*16+13].y = 0;
    
    out[idx*16+14].x = (double) ((in[idx] >> 28) & 0x3);
    out[idx*16+14].y = 0;
    
    out[idx*16+15].x = (double) ((in[idx] >> 30) & 0x3);
    out[idx*16+15].y = 0;
}

__global__ void
complex_to_complex_bitreverse(hipDoubleComplex* __restrict__ out,
                              int bitlen)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int rev_idx = (__brev(idx) >> (32-bitlen));
    if (rev_idx < idx)
    {
        hipDoubleComplex tmp = out[rev_idx];
        out[rev_idx] = out[idx];
        out[idx] = tmp;
    }
}

// Found at https://devtalk.nvidia.com/default/topic/814159/additional-cucomplex-functions-cucnorm-cucsqrt-cucexp-and-some-complex-double-functions-/
/*__host__ __device__ static __inline__ hipDoubleComplex
cuCexp(hipDoubleComplex x)
{
	double factor = exp(x.x);
	return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}*/

__global__ void
cooley_tukey_complex_fft(hipDoubleComplex* __restrict__ A,
                         int s,
                         int exp_sign,
                         hipDoubleComplex wn,
                         int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int m = (1 << s);
    int k = idx / (m/2);
    k *= m;
    int j = idx % (m/2);
    hipDoubleComplex w = make_hipDoubleComplex(cos(wn.y*j), sin(wn.y*j));
    hipDoubleComplex t, u;
    
    t = hipCmul(w, A[k + j + m/2]);
    u = A[k + j];
    
    A[k + j] = hipCadd(u, t);
    A[j + k + m/2] = hipCsub(u, t);
    
    if (m == N && exp_sign == 1)
    {
        A[k + j] = hipCdiv(A[k + j], make_hipDoubleComplex((double)N, 0));
        A[k + j + m/2] = hipCdiv(A[k + j + m/2], make_hipDoubleComplex((double)N, 0));
    }
}

__global__ void
pointwise_square(hipDoubleComplex* __restrict__ A)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    A[idx] = hipCmul(A[idx], A[idx]);
}

void
cooley_tukey_fft(hipDoubleComplex* a, int len)
{
    assert(isPow2(len));
    
    complex_to_complex_bitreverse<<<(len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, log2(len));

    
    for (int s = 1; s <= log2(len); s++)
    {
        cooley_tukey_complex_fft<<<((len/2)/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, s, -1, make_hipDoubleComplex(0, ((double)-2.0) * M_PI / (1<<s)), len);
    }
}

void
cooley_tukey_ifft(hipDoubleComplex* a, int len)
{
    assert(isPow2(len));
    
    complex_to_complex_bitreverse<<<(len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, log2(len));

    
    for (int s = 1; s <= log2(len); s++)
    {
        cooley_tukey_complex_fft<<<((len/2)/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, s, 1, make_hipDoubleComplex(0, ((double)2.0) * M_PI / (1<<s)), len);
    }
    
    /*
    for (int i = 0; i < len; i++)
    {
        out[i] = (unsigned int) (device_out[i].x + .5);
    }*/
}

__global__ void
cuda_combine8(hipDoubleComplex* a, unsigned int* c, unsigned long long* carry)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    unsigned long long result = 0;
    unsigned int w1 = (unsigned int) (a[idx*4].x + .5);
    unsigned int w2 = (unsigned int) (a[idx*4+1].x + .5);
    unsigned int w3 = (unsigned int) (a[idx*4+2].x + .5);
    unsigned int w4 = (unsigned int) (a[idx*4+3].x + .5);
    
    result = w4;
    result <<= 8;
    result += w3;
    result <<= 8;
    result += w2;
    result <<= 8;
    result += w1;
    
    c[idx] = result & 0xffffffff;
    carry[idx] = (result >> 32);
}

__global__ void
cuda_combine4(hipDoubleComplex* a, unsigned int* c, unsigned long long* carry)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    unsigned long long result = 0;
    unsigned int w1 = (unsigned int) (a[idx*8].x + .5);
    unsigned int w2 = (unsigned int) (a[idx*8+1].x + .5);
    unsigned int w3 = (unsigned int) (a[idx*8+2].x + .5);
    unsigned int w4 = (unsigned int) (a[idx*8+3].x + .5);
    unsigned int w5 = (unsigned int) (a[idx*8+4].x + .5);
    unsigned int w6 = (unsigned int) (a[idx*8+5].x + .5);
    unsigned int w7 = (unsigned int) (a[idx*8+6].x + .5);
    unsigned int w8 = (unsigned int) (a[idx*8+7].x + .5);
    
    result = w8;
    result <<= 4;
    result += w7;
    result <<= 4;
    result += w6;
    result <<= 4;
    result += w5;
    result <<= 4;
    result += w4;
    result <<= 4;
    result += w3;
    result <<= 4;
    result += w2;
    result <<= 4;
    result += w1;
    
    c[idx] = result & 0xffffffff;
    carry[idx] = (result >> 32);
}

__global__ void
cuda_combine2(hipDoubleComplex* a, unsigned int* c, unsigned long long* carry)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    unsigned long long result = 0;
    unsigned int w1 = (unsigned int) (a[idx*16].x + .5);
    unsigned int w2 = (unsigned int) (a[idx*16+1].x + .5);
    unsigned int w3 = (unsigned int) (a[idx*16+2].x + .5);
    unsigned int w4 = (unsigned int) (a[idx*16+3].x + .5);
    unsigned int w5 = (unsigned int) (a[idx*16+4].x + .5);
    unsigned int w6 = (unsigned int) (a[idx*16+5].x + .5);
    unsigned int w7 = (unsigned int) (a[idx*16+6].x + .5);
    unsigned int w8 = (unsigned int) (a[idx*16+7].x + .5);
    unsigned int w9 = (unsigned int) (a[idx*16+8].x + .5);
    unsigned int w10 = (unsigned int) (a[idx*16+9].x + .5);
    unsigned int w11 = (unsigned int) (a[idx*16+10].x + .5);
    unsigned int w12 = (unsigned int) (a[idx*16+11].x + .5);
    unsigned int w13 = (unsigned int) (a[idx*16+12].x + .5);
    unsigned int w14 = (unsigned int) (a[idx*16+13].x + .5);
    unsigned int w15 = (unsigned int) (a[idx*16+14].x + .5);
    unsigned int w16 = (unsigned int) (a[idx*16+15].x + .5);
    
    result = w16;
    result <<= 2;
    result += w15;
    result <<= 2;
    result += w14;
    result <<= 2;
    result += w13;
    result <<= 2;
    result += w12;
    result <<= 2;
    result += w11;
    result <<= 2;
    result += w10;
    result <<= 2;
    result += w9;
    result <<= 2;
    result += w8;
    result <<= 2;
    result += w7;
    result <<= 2;
    result += w6;
    result <<= 2;
    result += w5;
    result <<= 2;
    result += w4;
    result <<= 2;
    result += w3;
    result <<= 2;
    result += w2;
    result <<= 2;
    result += w1;
    
    c[idx] = result & 0xffffffff;
    carry[idx] = (result >> 32);
}

void
combine8(hipDoubleComplex* a, CudaBigInt& c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    cuda_combine8<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, c.val, long_carry);
    
    cuda_long_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    
    
    cuda_malloc_free(long_carry);
    cuda_malloc_free(byte_carry1);
    cuda_malloc_free(byte_carry2);
    cuda_malloc_free(should_carry_cuda);
}

void
combine4(hipDoubleComplex* a, CudaBigInt& c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    cuda_combine4<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, c.val, long_carry);
    
    cuda_long_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    
    
    cuda_malloc_free(long_carry);
    cuda_malloc_free(byte_carry1);
    cuda_malloc_free(byte_carry2);
    cuda_malloc_free(should_carry_cuda);
}

void
combine2(hipDoubleComplex* a, CudaBigInt& c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    cuda_combine2<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, c.val, long_carry);
    
    cuda_long_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    
    
    cuda_malloc_free(long_carry);
    cuda_malloc_free(byte_carry1);
    cuda_malloc_free(byte_carry2);
    cuda_malloc_free(should_carry_cuda);
}

void
fft_square(CudaBigInt& a, CudaBigInt& c)
{
    hipDoubleComplex* cuda_a;
    
    cuda_malloc_clear((void**) &cuda_a, sizeof(*cuda_a)*a.word_len*32);
    
    split2<<<(a.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a.val, cuda_a);
    
    cooley_tukey_fft(cuda_a, a.word_len*32);
    pointwise_square<<<(a.word_len*32/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(cuda_a);
    cooley_tukey_ifft(cuda_a, a.word_len*32);
    
    combine2(cuda_a, c);
    
    cuda_malloc_free(cuda_a);
}

