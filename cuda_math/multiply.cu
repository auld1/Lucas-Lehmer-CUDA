#include "hip/hip_runtime.h"
#include "multiply.h"

#include "bigint.h"
#include "carry.h"
#include "memory.h"

#include <assert.h>

__device__ unsigned int
cuda_multiply_gradeschool_digit(const unsigned int* __restrict__ a,
                                const unsigned int* __restrict__ b,
                                int digit,
                                unsigned long long * __restrict__ carry_out,
                                unsigned int carry_in,
                                const int N)
{
    unsigned long long carry = 0;
    unsigned int temp = 0;
    unsigned int result = 0;
    int a_i = 0;
    int b_i = 0;

    result = carry_in;
    for (b_i = max(0, digit - N); b_i <= min(digit, N); b_i++)
    {
        a_i = digit - b_i;
        temp = a[a_i] * b[b_i];
        result += temp;
        if (result < temp)
        {
            carry++;
        }
        carry += __umulhi(a[a_i], b[b_i]);
    }
    *carry_out = carry;
    return result;
}

__global__ void
cuda_multiply_gradeschool(const unsigned int* __restrict__ a,
                          const unsigned int* __restrict__ b,
                          unsigned int* __restrict__ c,
                          unsigned long long* __restrict__ carry_out,
                          const int N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    c[i] = cuda_multiply_gradeschool_digit(a, b, i, &carry_out[i], 0, N);
}

void
multiply(CudaBigInt a, CudaBigInt b, CudaBigInt c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    assert(a.word_len * b.word_len == c.word_len);
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
     
    cuda_multiply_gradeschool<<<64, c.word_len/64>>>(a.val, b.val, c.val, long_carry, a.word_len);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    cuda_long_carry<<<64, c.word_len/64>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<64, c.word_len/64>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    c.sign = a.sign*b.sign;
}
