#include "hip/hip_runtime.h"
#include "multiply.h"

#include "bigint.h"
#include "carry.h"
#include "memory.h"

#include <assert.h>
#include <stdio.h>

__device__ unsigned int
cuda_multiply_gradeschool_digit(const unsigned int* __restrict__ a,
                                const unsigned int* __restrict__ b,
                                int digit,
                                unsigned long long * __restrict__ carry_out,
                                unsigned int carry_in,
                                const int N)
{
    unsigned long long carry = 0;
    unsigned int temp = 0;
    unsigned int result = 0;
    int a_i = 0;
    int b_i = 0;

    result = carry_in;
    for (b_i = max(0, digit - N/2 + 1); b_i <= min(digit, N/2 - 1); b_i++)
    {
        a_i = digit - b_i;
        temp = a[a_i] * b[b_i];
        result += temp;
        if (result < temp)
        {
            carry++;
        }
        carry += __umulhi(a[a_i], b[b_i]);
    }
    *carry_out = carry;
    return result;
}

__global__ void
cuda_multiply_gradeschool(const unsigned int* __restrict__ a,
                          const unsigned int* __restrict__ b,
                          unsigned int* __restrict__ c,
                          unsigned long long* __restrict__ carry_out,
                          const int N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    c[i] = cuda_multiply_gradeschool_digit(a, b, i, &carry_out[i], 0, N);
}

void
multiply(CudaBigInt a, CudaBigInt b, CudaBigInt c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    assert(a.word_len == b.word_len);
    assert(a.word_len + b.word_len == c.word_len);
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
     
    cuda_multiply_gradeschool<<<64, c.word_len/64>>>(a.val, b.val, c.val, long_carry, c.word_len);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    cuda_long_carry<<<64, c.word_len/64>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<64, c.word_len/64>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    c.sign = a.sign*b.sign;
}

int
main()
{
    CudaBigInt a;
    CudaBigInt b;
    CudaBigInt c(4096);
    
    unsigned int a_host[a.word_len];
    unsigned int b_host[b.word_len];
    unsigned int c_host[c.word_len];
    
    
    int i = 0;
    
    for(i = 0; i < c.word_len; i++)
    {
        c_host[i] = 0;
    }
    
    for(i = 0; i < a.word_len; i++)
    {
        a_host[i] = 0;
        b_host[i] = 0;
    }
    
    for(i = 0; i < a.word_len; i++)
    {
        a_host[i] = 0xffffffff;
        b_host[i] = 0xffffffff;
    }
    
    
    for(i = 0; i < c.word_len; i++)
    {
        c_host[i] = 0;
    }
    
    
    hipMemcpy(a.val, a_host, a.word_len * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(b.val, b_host, b.word_len * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(c.val, c_host, c.word_len * sizeof(unsigned int), hipMemcpyHostToDevice);
    
    multiply(a, b, c);
    
    
    hipMemcpy(c_host, c.val, c.word_len * sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    
    for(i = 0; i < c.word_len; i++)
    {
        printf("%x\n", c_host[i]);
    }
    

    return 0;
}
