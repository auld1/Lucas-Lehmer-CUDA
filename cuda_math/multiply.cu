#include "hip/hip_runtime.h"
#include "multiply.h"

#include "bigint.h"
#include "carry.h"
#include "memory.h"
#include "rand.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmp.h>

#define MULTIPLY_BLOCK_SIZE (64)

void
set_mpz_uint(mpz_t t, unsigned int* val, int len)
{
    mpz_import(t, len, -1, sizeof(unsigned int), -1, 0, val);
}

__device__ unsigned int
cuda_multiply_gradeschool_digit(const unsigned int* __restrict__ a,
                                const unsigned int* __restrict__ b,
                                int digit,
                                unsigned long long * __restrict__ carry_out,
                                unsigned long long carry_in,
                                const int N)
{
    unsigned long long carry = 0;
    unsigned int temp = 0;
    unsigned int result = 0;
    int a_i = 0;
    int b_i = 0;

    result = carry_in & 0xffffffff;
    for (b_i = max(0, digit - N/2 + 1); b_i <= min(digit, N/2 - 1); b_i++)
    {
        a_i = digit - b_i;
        temp = a[a_i] * b[b_i];
        result += temp;
        if (result < temp)
        {
            carry++;
        }
        carry += __umulhi(a[a_i], b[b_i]);
    }
    carry += (carry_in >> 32) & 0xffffffff;
    *carry_out = carry;
    return result;
}

__global__ void
cuda_multiply_gradeschool(const unsigned int* __restrict__ a,
                          const unsigned int* __restrict__ b,
                          unsigned int* __restrict__ c,
                          unsigned long long* __restrict__ carry_out,
                          const int N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    c[i] = cuda_multiply_gradeschool_digit(a, b, i, &carry_out[i], 0, N);
}

void
multiply(CudaBigInt& a, CudaBigInt& b, CudaBigInt& c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    assert(a.word_len == b.word_len);
    assert(a.word_len + b.word_len == c.word_len);
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
     
    cuda_multiply_gradeschool<<<(c.word_len/MULTIPLY_BLOCK_SIZE), MULTIPLY_BLOCK_SIZE>>>(a.val, b.val, c.val, long_carry, c.word_len);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    cuda_long_carry<<<(c.word_len/MULTIPLY_BLOCK_SIZE), MULTIPLY_BLOCK_SIZE>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<(c.word_len/MULTIPLY_BLOCK_SIZE), MULTIPLY_BLOCK_SIZE>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    c.sign = a.sign*b.sign;
    
    hipFree(long_carry);
    hipFree(byte_carry1);
    hipFree(byte_carry2);
    hipFree(should_carry_cuda);
}

int
test()
{
    CudaBigInt a(1024*1024*4);
    CudaBigInt b(1024*1024*4);
    CudaBigInt c(1024*1024*4*2);
    
    mpz_t a_gmp;
    mpz_t b_gmp;
    mpz_t c_gmp;
    mpz_t mul_gmp;
    
    mpz_init2(a_gmp, a.word_len*32);
    mpz_init2(b_gmp, b.word_len*32);
    mpz_init2(c_gmp, c.word_len*32);
    mpz_init2(mul_gmp, c.word_len*32);
    
    unsigned int* a_host;
    unsigned int* b_host;
    unsigned int c_host[c.word_len];
    
    srand(time(NULL));
    
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    
    a_host = get_random_array(a.val, a.word_len);
    b_host = get_random_array(b.val, a.word_len);
    
    set_mpz_uint(a_gmp, a_host, a.word_len);
    set_mpz_uint(b_gmp, b_host, b.word_len);
    
    multiply(a, b, c);
    
    hipMemcpy(c_host, c.val, c.word_len * sizeof(unsigned int), hipMemcpyDeviceToHost);
    set_mpz_uint(c_gmp, c_host, c.word_len);
    mpz_mul(mul_gmp, a_gmp, b_gmp);
    
    assert(0 == mpz_cmp(mul_gmp, c_gmp));
    
    return 0;
}


int
main(void)
{
    int i = 0;
    printf("Testing 1000 iterations of multiply on random digits\n");
    for (i = 0; i < 1000; i++)
    {
        test();
    }
    printf("Passed\n");
}
