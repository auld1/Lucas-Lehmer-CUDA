#include "hip/hip_runtime.h"
#include "carry.h"

#include <stdbool.h>

__global__
void cuda_long_carry(unsigned int*   c,
                     const unsigned long long*   carry_in,
                     unsigned char*   carry_out,
                     bool*   needs_carry)
{
    int c_i = blockIdx.x*blockDim.x + threadIdx.x;
    
    carry_out[c_i] = 0;
    if (c_i - 1 >= 0)
    {
        unsigned int temp = carry_in[c_i - 1] & 0xffffffff;
        c[c_i] += temp;
        if (c[c_i] < temp)
        {
            carry_out[c_i]++;
            *needs_carry = true;
        }
    }
    
    if (c_i - 2 >= 0)
    {
        unsigned int temp = (carry_in[c_i - 2] >> 32) & 0xffffffff;
        c[c_i] += temp;
        if (c[c_i] < temp)
        {
            carry_out[c_i]++;
            *needs_carry = true;
        }
    }
}

__global__
void cuda_int_carry(unsigned int*   c,
                    const unsigned int*   carry_in,
                    unsigned char*   carry_out,
                    bool*   needs_carry)
{
    int c_i = blockIdx.x*blockDim.x + threadIdx.x;
    
    carry_out[c_i] = 0;
    if (c_i - 1 >= 0)
    {
        c[c_i] += carry_in[c_i - 1];
        if (c[c_i] < carry_in[c_i - 1])
        {
            carry_out[c_i]++;
            *needs_carry = true;
        }
    }
}

__global__
void cuda_byte_carry(unsigned int*   c,
                     const unsigned char*   carry_in,
                     unsigned char*   carry_out,
                     bool*   needs_carry)
{
    int c_i = blockIdx.x*blockDim.x + threadIdx.x;
    
    carry_out[c_i] = 0;
    if (c_i - 1 >= 0)
    {
        unsigned int temp = c[c_i];
        c[c_i] += carry_in[c_i - 1];
        if (c[c_i] < temp)
        {
            carry_out[c_i] = 1;
            *needs_carry = true;
        }
    }
}

__global__
void cuda_negative_byte_carry(unsigned int*   c,
                              const unsigned char*   carry_in,
                              unsigned char*   carry_out,
                              bool*   needs_carry)
{
    int c_i = blockIdx.x*blockDim.x + threadIdx.x;
    
    carry_out[c_i] = 0;
    if (c_i - 1 >= 0)
    {
        unsigned int temp = c[c_i];
        c[c_i] -= carry_in[c_i - 1];
        if (c[c_i] > temp)
        {
            carry_out[c_i] = 1;
            *needs_carry = true;
        }
    }
}
