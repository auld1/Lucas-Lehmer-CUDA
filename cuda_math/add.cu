#include "hip/hip_runtime.h"
#include "add.h"

#include "bigint.h"
#include "carry.h"
#include "memory.h"

#include <assert.h>

__device__
unsigned int cuda_add_digit(const unsigned int* __restrict__ a,
                            const unsigned int* __restrict__ b,
                            int digit,
                            unsigned char* __restrict__ carry_out,
                            unsigned int carry_in,
                            const int N)
{
    unsigned char carry = 0;
    unsigned int result = 0;
    unsigned int temp = 0;
    
    result = carry_in;
    temp = a[digit] + b[digit];
    if (temp < a[digit])
    {
        carry++;
    }
    
    result += temp;
    if (result < temp)
    {
        carry++;
    }
    
    *carry_out = carry;
    return result;
}

__global__
void cuda_add(const unsigned int* __restrict__ a,
              const unsigned int* __restrict__ b,
              unsigned int* __restrict__ c,
              unsigned char* __restrict__ carry_out,
              const int N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    c[i] = cuda_add_digit(a, b, i, &carry_out[i], 0, N);
}

void add(CudaBigInt a, CudaBigInt b, CudaBigInt c)
{
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    
    if (a.sign == -1)
    {
        if (b.sign == 1)
        {
            //subtract(b, a, c);
            return;
        }
        // Both signs are -1, we will add them together as positives
        // but change the sign of c
        c.sign = -1;
    } else if (b.sign == -1)
    {
        //subtract(a, b, c);
        return;
    }
    
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
     
    cuda_add<<<64, c.word_len/64>>>(a.val, b.val, c.val, byte_carry1, c.word_len);
    err = hipDeviceSynchronize();
    assert(err == hipSuccess);
    
    do
    {
        cuda_byte_carry<<<64, c.word_len/64>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    } while (should_carry_host);
    
}
