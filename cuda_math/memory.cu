
#include <hip/hip_runtime.h>
#include "memory.h"

#include <assert.h>
#include <stdlib.h>
#include <stdio.h>

#define CACHED


#ifdef CACHED
#define CACHELEN 128

typedef struct _tup
{
    size_t bytes;
    void* ptr;
    bool free;
} _tup;

_tup cache[CACHELEN];
bool initialized = false;
#endif // CACHED


void
cuda_malloc_clear(void** ptr, size_t bytes)
{
    hipError_t err;
    
#ifdef CACHED
    if (!initialized)
    {
        for (int i = 0; i < CACHELEN; i++)
        {
            cache[i].bytes = 0;
            cache[i].ptr = 0;
            cache[i].free = true;
        }
        initialized = true;
    }
    
    for (int i = 0; i < CACHELEN; i++)
    {
        if (cache[i].free && cache[i].bytes == bytes)
        {
            // We don't have to remalloc, we already have a valid free ptr
            cache[i].free = false;
            *ptr = cache[i].ptr;
            err = hipMemset(*ptr, 0, bytes);
            assert(err == hipSuccess);
            return;
        }
    }
#endif // CACHED

    // Malloc to device, check for errors
    err = hipMalloc(ptr, bytes);
    assert(err == hipSuccess);

    // Set val to 0, check for errors
    err = hipMemset(*ptr, 0, bytes);
    assert(err == hipSuccess);

#ifdef CACHED
    for (int i = 0; i < CACHELEN; i++)
    {
        if (cache[i].free && cache[i].ptr == 0)
        {
            cache[i].free = false;
            cache[i].ptr = *ptr;
            cache[i].bytes = bytes;
            return;
        }
    }
#endif // CACHED
}

void
cuda_malloc_free(void* ptr)
{
#ifdef CACHED
    for (int i = 0; i < CACHELEN; i++)
    {
        if (cache[i].ptr == ptr)
        {
            cache[i].free = true;
            return;
        }
    }
#endif // CACHED

    hipFree(ptr);
}
