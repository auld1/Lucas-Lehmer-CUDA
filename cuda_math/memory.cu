
#include <hip/hip_runtime.h>
#include "memory.h"

#include <assert.h>

void
cuda_malloc_clear(void** ptr, size_t bytes)
{
    hipError_t err;
    
    // Malloc to device, check for errors
    err = hipMalloc(ptr, bytes);
    assert(err == hipSuccess);

    // Set val to 0, check for errors
    err = hipMemset(*ptr, 0, bytes);
    assert(err == hipSuccess);
}
