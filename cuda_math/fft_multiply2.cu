#include "hip/hip_runtime.h"
#include "fft_multiply.h"

#include "bigint.h"
#include "carry.h"
#include "memory.h"

#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>


#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>


#define FFT_BLOCK_SIZE (128)

#define FFT_USE_DOUBLE

#ifdef FFT_USE_DOUBLE

#define BITS_PER_FLOAT 8
#define hipComplex hipDoubleComplex
#define make_hipComplex make_hipDoubleComplex
#define floatType double
#define cuAdd hipCadd
#define cuSub hipCsub
#define cuMul hipCmul
#define cuDiv hipCdiv

#else

#define BITS_PER_FLOAT 4
#define hipComplex hipFloatComplex
#define make_hipComplex make_hipFloatComplex
#define floatType float
#define cuAdd hipCaddf
#define cuSub hipCsubf
#define cuMul hipCmulf
#define cuDiv hipCdivf

#endif


#define FLOATS_PER_WORD (32/BITS_PER_FLOAT)


__global__ void
split(const unsigned int* __restrict__ in,
      hipComplex* __restrict__ out)
{
    int idx = (blockIdx.x*blockDim.x + threadIdx.x);
    
    for(int i = 0; i < FLOATS_PER_WORD; i++)
    {
        out[idx*FLOATS_PER_WORD+i].x = (floatType) ((in[idx] >> (i*BITS_PER_FLOAT)) & ((1<<BITS_PER_FLOAT) - 1));
        out[idx*FLOATS_PER_WORD+i].y = 0;
    }
}

__global__ void
complex_to_complex_bitreverse(hipComplex* __restrict__ out,
                              int bitlen)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int rev_idx = (__brev(idx) >> (32-bitlen));
    if (rev_idx < idx)
    {
        hipComplex tmp = out[rev_idx];
        out[rev_idx] = out[idx];
        out[idx] = tmp;
    }
}

__global__ void
cooley_tukey_complex_fft(hipComplex* __restrict__ A,
                         int s,
                         int exp_sign,
                         hipComplex wn,
                         int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int m = (1 << s);
    int k = idx / (m/2);
    k *= m;
    int j = idx % (m/2);
    hipComplex w = make_hipComplex(cos(wn.y*j), sin(wn.y*j));
    hipComplex t, u;
    
    t = cuMul(w, A[k + j + m/2]);
    u = A[k + j];
    
    A[k + j] = cuAdd(u, t);
    A[j + k + m/2] = cuSub(u, t);
    
    if (m == N && exp_sign == 1)
    {
        A[k + j] = cuDiv(A[k + j], make_hipComplex((floatType)N, 0));
        A[k + j + m/2] = cuDiv(A[k + j + m/2], make_hipComplex((floatType)N, 0));
    }
}

__global__ void
pointwise_square(hipComplex* __restrict__ A)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    A[idx] = cuMul(A[idx], A[idx]);
}


void
cooley_tukey_fft(hipComplex* a, int len)
{
    assert(isPow2(len));
    
    complex_to_complex_bitreverse<<<(len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, log2(len));

    
    for (int s = 1; s <= log2(len); s++)
    {
        cooley_tukey_complex_fft<<<((len/2)/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, s, -1, make_hipComplex(0, ((floatType)-2.0) * M_PI / (1<<s)), len);
    }
}

void
cooley_tukey_ifft(hipComplex* a, int len)
{
    assert(isPow2(len));
    
    complex_to_complex_bitreverse<<<(len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, log2(len));

    
    for (int s = 1; s <= log2(len); s++)
    {
        cooley_tukey_complex_fft<<<((len/2)/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, s, 1, make_hipComplex(0, ((floatType)2.0) * M_PI / (1<<s)), len);
    }
}

__global__ void
cuda_combine(hipComplex* a, unsigned int* c, unsigned long long* carry)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    unsigned long long result = 0;
    /*
    unsigned int w1 = (unsigned int) (a[idx*4].x + .5);
    unsigned int w2 = (unsigned int) (a[idx*4+1].x + .5);
    unsigned int w3 = (unsigned int) (a[idx*4+2].x + .5);
    unsigned int w4 = (unsigned int) (a[idx*4+3].x + .5);
    
    result = w4;
    result <<= 8;
    result += w3;
    result <<= 8;
    result += w2;
    result <<= 8;
    result += w1;
    */
    
    for (int i = FLOATS_PER_WORD-1; i >= 0; i--)
    {
        result <<= BITS_PER_FLOAT;
        result += (a[idx*FLOATS_PER_WORD+i].x + .5);
    }
    c[idx] = result & 0xffffffff;
    carry[idx] = (result >> 32);
    
}

void
combine(hipComplex* a, CudaBigInt& c)
{
    unsigned long long* long_carry;
    unsigned char* byte_carry1;
    unsigned char* byte_carry2;
    bool* should_carry_cuda;
    bool should_carry_host;
    hipError_t err;
    
    cuda_malloc_clear((void**) &long_carry, c.word_len * sizeof(*long_carry));
    cuda_malloc_clear((void**) &byte_carry1, c.word_len * sizeof(*byte_carry1));
    cuda_malloc_clear((void**) &byte_carry2, c.word_len * sizeof(*byte_carry2));
    cuda_malloc_clear((void**) &should_carry_cuda, sizeof(bool));
    
    cuda_combine<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a, c.val, long_carry);
    
    cuda_long_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, long_carry, byte_carry1, should_carry_cuda);
    
    err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    err = hipMemset(should_carry_cuda, 0, sizeof(bool));
    assert(err == hipSuccess);
    
    while (should_carry_host)
    {
        cuda_byte_carry<<<(c.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(c.val, byte_carry1, byte_carry2, should_carry_cuda);
    
        err = hipMemcpy(&should_carry_host, should_carry_cuda, sizeof(bool), hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
        
        err = hipMemset(should_carry_cuda, 0, sizeof(bool));
        assert(err == hipSuccess);
        
        unsigned char* temp = byte_carry1;
        byte_carry1 = byte_carry2;
        byte_carry2 = temp;
    }
    
    
    cuda_malloc_free(long_carry);
    cuda_malloc_free(byte_carry1);
    cuda_malloc_free(byte_carry2);
    cuda_malloc_free(should_carry_cuda);
}


void
fft_square(CudaBigInt& a, CudaBigInt& c)
{
    hipComplex* cuda_a;
    
    cuda_malloc_clear((void**) &cuda_a, sizeof(*cuda_a)*a.word_len*FLOATS_PER_WORD*2);
    
    split<<<(a.word_len/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(a.val, cuda_a);
    
    cooley_tukey_fft(cuda_a, a.word_len*FLOATS_PER_WORD*2);
    pointwise_square<<<(a.word_len*FLOATS_PER_WORD*2/FFT_BLOCK_SIZE), FFT_BLOCK_SIZE>>>(cuda_a);
    cooley_tukey_ifft(cuda_a, a.word_len*FLOATS_PER_WORD*2);
    
    combine(cuda_a, c);
    
    cuda_malloc_free(cuda_a);
}
